// Copyright (C) 2023 Alan Jian (alanjian85@outlook.com)
// SPDX-License-Identifier: MIT

#include <config/types.h>
#include <core/film.hpp>
#include <core/utils.h>
#include <shapes/sphere.hpp>

const int tileSize = 16;

PRISM_KERNEL void render(prism::film film) {
    int nTilesX = (film.width + tileSize - 1) / tileSize;
    int x = blockIdx.x % nTilesX * tileSize + threadIdx.x % tileSize;
    int y = blockIdx.x / nTilesX * tileSize + threadIdx.x / tileSize;
    float u = static_cast<real_t>(x) / (film.width - 1);
    float v = static_cast<real_t>(y) / (film.height - 1);
    prism::sphere sphere(prism::point3f(0, 0, 1), 0.5);
    prism::ray ray(prism::point3f(u - 0.5, v - 0.5, 0),
                   prism::vector3f(0, 0, 1));
    if (sphere.intersect(ray)) {
        film.add_sample(prism::point2i(x, y), prism::color(u, v, 0.25));
    } else {
        film.add_sample(prism::point2i(x, y), prism::color(0, 0, 0));
    }
}

int main() {
    prism::film film(256, 256);
    int nTiles = ((film.width + tileSize - 1) / tileSize) *
                 ((film.height + tileSize - 1) / tileSize);
    render<<<nTiles, tileSize * tileSize>>>(film);
    hipDeviceSynchronize();
    film.write_image("image.png");
    film.free();
    return 0;
}
