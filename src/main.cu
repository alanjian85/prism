// Copyright (C) 2023 Alan Jian (alanjian85@outlook.com)
// SPDX-License-Identifier: MIT

#include <cameras/persp_camera.hpp>
#include <config/types.h>
#include <core/film.hpp>
#include <core/utils.h>
#include <shapes/sphere.hpp>

const int tileSize = 16;

PRISM_KERNEL void render(prism::persp_camera &camera) {
    int nTilesX = (camera.film.width + tileSize - 1) / tileSize;
    int x = blockIdx.x % nTilesX * tileSize + threadIdx.x % tileSize;
    int y = blockIdx.x / nTilesX * tileSize + threadIdx.x / tileSize;
    prism::sphere sphere(prism::point3f(0, 0, 2), 0.5);
    prism::ray ray = camera.generate_ray(prism::point2i(x, y));
    if (sphere.intersect(ray)) {
        camera.film.add_sample(prism::point2i(x, y), prism::color(1, 1, 1));
    } else {
        camera.film.add_sample(prism::point2i(x, y), prism::color(0, 0, 0));
    }
}

int main() {
    prism::persp_camera *camera;
    hipMallocManaged(&camera, sizeof(prism::persp_camera));
    new (camera) prism::persp_camera(256, 256);
    camera->o = prism::point3f(0, 0, 0);
    camera->d = prism::vector3f(0, 0, 1);
    camera->near = 1;
    camera->far = 1000;
    int nTiles = ((camera->film.width + tileSize - 1) / tileSize) *
                 ((camera->film.height + tileSize - 1) / tileSize);
    render<<<nTiles, tileSize * tileSize>>>(*camera);
    hipDeviceSynchronize();
    camera->film.write_image("image.png");
    hipFree(camera);
    return 0;
}
