#include "hip/hip_runtime.h"
// Copyright (C) 2023 Alan Jian (alanjian85@outlook.com)
// SPDX-License-Identifier: MIT

#include <iostream>
#include <memory>

#include <tiny_obj_loader.h>

#include "core/camera.hpp"
#include "core/utils.h"
#include "scene/scene.hpp"
const int tileSize = 16;

PRISM_KERNEL void render(Camera &camera, Scene &scene) {
    int nTilesX = (camera.film.width() + tileSize - 1) / tileSize;
    int x = blockIdx.x % nTilesX * tileSize + threadIdx.x % tileSize;
    int y = blockIdx.x / nTilesX * tileSize + threadIdx.x / tileSize;
    Real u = static_cast<Real>(x) / (camera.film.width() - 1);
    Real v = static_cast<Real>(y) / (camera.film.height() - 1);
    Ray ray = camera.generateRay(Point2f(u, 1 - v));
    Interaction interaction;
    scene.intersect(ray, interaction);
    if (dot(ray.d, interaction.n) > 0)
        interaction.n = -interaction.n;
    Real attenuation = 1 / (camera.o - interaction.p).lengthSquared();
    Vector3f lightDir = normalize(camera.o - interaction.p);
    Vector3f halfwayDir = lightDir;
    Real diffuse = dot(interaction.n, lightDir);
    Real specular = pow(dot(interaction.n, halfwayDir), 32);
    Color color = (diffuse * normalToColor(interaction.n) + specular * Color(1, 1, 1)) * attenuation;
    color = clamp(color, Color(0), Color(1));
    camera.film.addSample(Point2f(u, v), color);
}

int main() {
    tinyobj::ObjReader reader;
    reader.ParseFromFile("viking_room.obj");

    std::vector<Triangle> primitives;
    auto& attrib = reader.GetAttrib();
    auto& shapes = reader.GetShapes();
    for (size_t s = 0; s < shapes.size(); s++) {
        size_t index_offset = 0;
        for (size_t f = 0; f < shapes[s].mesh.num_face_vertices.size(); f++) {
            tinyobj::index_t idx1 = shapes[s].mesh.indices[index_offset];
            tinyobj::real_t x1 = attrib.vertices[3 * size_t(idx1.vertex_index) + 0];
            tinyobj::real_t y1 = attrib.vertices[3 * size_t(idx1.vertex_index) + 1];
            tinyobj::real_t z1 = attrib.vertices[3 * size_t(idx1.vertex_index) + 2];
            ++index_offset;

            tinyobj::index_t idx2 = shapes[s].mesh.indices[index_offset];
            tinyobj::real_t x2 = attrib.vertices[3 * size_t(idx2.vertex_index) + 0];
            tinyobj::real_t y2 = attrib.vertices[3 * size_t(idx2.vertex_index) + 1];
            tinyobj::real_t z2 = attrib.vertices[3 * size_t(idx2.vertex_index) + 2];
            ++index_offset;

            tinyobj::index_t idx3 = shapes[s].mesh.indices[index_offset];
            tinyobj::real_t x3 = attrib.vertices[3 * size_t(idx3.vertex_index) + 0];
            tinyobj::real_t y3 = attrib.vertices[3 * size_t(idx3.vertex_index) + 1];
            tinyobj::real_t z3 = attrib.vertices[3 * size_t(idx3.vertex_index) + 2];
            ++index_offset;

            primitives.push_back(Triangle(Vector3f(x1, y1, z1),
                                          Vector3f(x2, y2, z2),
                                          Vector3f(x3, y3, z3)));
        }
    }
    auto scene = std::make_unique<Scene>(primitives);

    const int width = 1024, height = 1024;
    auto camera = std::make_unique<Camera>(width, height, CameraType::Persp,
                      Vector3f(1, 0, 1), Vector3f(-1, 0, -1), Vector3f(0, 0, 1),
                      radians(90));

    int nTiles = ((camera->film.width() + tileSize - 1) / tileSize) *
                 ((camera->film.height() + tileSize - 1) / tileSize);
    render<<<nTiles, tileSize * tileSize>>>(*camera, *scene);
    hipDeviceSynchronize();
    camera->film.writeImage("image.png");

    return 0;
}
