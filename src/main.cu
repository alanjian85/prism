// Copyright (C) 2023 Alan Jian (alanjian85@outlook.com)
// SPDX-License-Identifier: MIT

#include <thrust/device_new.h>

#include <cameras/persp_camera.hpp>
#include <config/types.h>
#include <core/film.hpp>
#include <core/utils.h>
#include <shapes/sphere.hpp>

const int tileSize = 16;

PRISM_KERNEL void construct_objects(prism::persp_camera *camera,
                      void *pixels, int width, int height)
{
    new (camera) prism::persp_camera(pixels, width, height); 
    camera->o = prism::point3f(0, 0, 0);
    camera->d = prism::vector3f(0, 0, 1);
    camera->near = 1;
    camera->far = 1000;
}

PRISM_KERNEL void render(prism::camera &camera) {
    int nTilesX = (camera.film.width + tileSize - 1) / tileSize;
    int x = blockIdx.x % nTilesX * tileSize + threadIdx.x % tileSize;
    int y = blockIdx.x / nTilesX * tileSize + threadIdx.x / tileSize;
    prism::sphere sphere(prism::point3f(0, 0, 2), 0.5);
    prism::ray ray = camera.generate_ray(prism::point2i(x, y));
    if (sphere.intersect(ray)) {
        camera.film.add_sample(prism::point2i(x, y), prism::color(1, 1, 1));
    } else {
        camera.film.add_sample(prism::point2i(x, y), prism::color(0, 0, 0));
    }
}

int main() {
    const int width = 256, height = 256;
    void *pixels;
    hipMallocManaged(&pixels, width * height * 3);
    prism::persp_camera *camera;
    hipMallocManaged(&camera, sizeof(prism::persp_camera));
    construct_objects<<<1, 1>>>(camera, pixels, width, height);
    hipDeviceSynchronize();
    int nTiles = ((camera->film.width + tileSize - 1) / tileSize) *
                 ((camera->film.height + tileSize - 1) / tileSize);
    render<<<nTiles, tileSize * tileSize>>>(*camera);
    hipDeviceSynchronize();
    camera->film.write_image("image.png");
    hipFree(camera);
    hipFree(pixels); 
    return 0;
}
